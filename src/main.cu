#include "hip/hip_runtime.h"
#include "cuda_try.cuh"
#include "cuda_time.cuh"
#include <cstdint>
#include <cstddef>
#include "fast_prng.cuh"

#define THREADS_PER_WARP 32
#define TABLE_SIZE ((size_t)THREADS_PER_WARP * sizeof(uint16_t))

struct block_stats {
    size_t idx;
    size_t size_c;
    size_t size_d;
    size_t count;
    size_t gridstride;
    size_t final_block_elem_count;
};

__device__ int get_warp_idx()
{
    return threadIdx.x / THREADS_PER_WARP;
}

__device__ int get_warp_offset()
{
    return threadIdx.x % THREADS_PER_WARP;
}

constexpr size_t sb_max_elems_c(size_t sb_elem_count)
{
    return sb_elem_count + (sb_elem_count + 7) / 8;
}

size_t __device__ __host__ get_compressed_size(size_t uncompressed_data_size, size_t elem_count)
{
    return sb_max_elems_c(elem_count) * THREADS_PER_WARP * sizeof(uint32_t) + TABLE_SIZE;
}

__device__ block_stats get_block_stats(size_t uncompressed_data_size, size_t elem_count)
{
    block_stats b;
    int warp_count = (blockDim.x / THREADS_PER_WARP);
    int warp_idx = (threadIdx.x / THREADS_PER_WARP);
    b.size_c = get_compressed_size(uncompressed_data_size, elem_count);
    b.size_d = elem_count * THREADS_PER_WARP * sizeof(uint32_t);
    b.count = (uncompressed_data_size + b.size_d - 1) / b.size_d;
    b.gridstride = gridDim.x * warp_count;
    b.idx = blockIdx.x * (blockDim.x / THREADS_PER_WARP) + warp_idx;
    if (b.count * b.size_d == uncompressed_data_size) {
        b.final_block_elem_count = elem_count * THREADS_PER_WARP;
    }
    else {
        b.final_block_elem_count = (uncompressed_data_size - (uncompressed_data_size / b.size_d * b.size_d)) / sizeof(uint32_t);
    }
    return b;
}

template <typename T> void bit_print(T data, bool spacing = true)
{
    size_t typewidth_m1 = sizeof(T) * 8 - 1;
    for (int i = typewidth_m1; i >= 0; i--) {
        printf("%c", (data >> i) & 0b1 ? '1' : '0');
        if (spacing && i < typewidth_m1 && i > 0 && i % 8 == 0) {
            printf(" ");
        }
    }
}

template <typename T> __host__ __device__ T encode_sign(T val, bool negative)
{
    return (val << 1) | (negative ? 0b1 : 0b0);
}

// returns the sign, AND shifts out the seign bit from the val
template <typename T> __host__ __device__ bool decode_sign(T* val)
{
    bool negative = (*val & 0b1);
    *val >>= 1;
    return negative;
}

#define SDIFF_VMAX (UINT32_MAX & 0b01111111111111111111111111111111)
//  value v2 succeeds v1, get the diff encoded with lsb sign
//  use this to encode val, given the base
template <typename T> __host__ __device__ T encoded_sign_diff(T base, T val)
{
    // encode using unsigned overflows if going from very large to very small (or other way), benefits for a few cases
    if (SDIFF_VMAX - base + val <= (val > base ? val - base : base - val)) {
        return encode_sign(SDIFF_VMAX - base + val + 1, false);
    }
    if (SDIFF_VMAX - val + base <= (val > base ? val - base : base - val)) {
        return encode_sign(SDIFF_VMAX - val + base + 1, true);
    }
    // normal encoding
    if (val > base) {
        return encode_sign(val - base, false);
    }
    return encode_sign(base - val, true);
}

// use this to decode the value underlying signdiff, given the base
template <typename T> T __host__ __device__ decode_sign_diff(T base, T signdiff)
{
    bool negative_diff = decode_sign(&signdiff);
    if (negative_diff) {
        return base - signdiff;
    }
    return base + signdiff;
}

// TODO remove byte splicing from compression

template <size_t SB_ELEM_COUNT, bool USE_DIFFERENTIAL_PRE_ENCODING> __host__ __device__ size_t compress_block(uint32_t* data_d, uint8_t* data_c)
{
    size_t in_count = SB_ELEM_COUNT;
    size_t in_idx = 0;
    size_t out_byte_idx = 0;
    while (in_idx < in_count) {
        uint64_t working = 0;
        uint64_t inel2[2];
        if (USE_DIFFERENTIAL_PRE_ENCODING) {
            // currently this can only handle elements that are not using the most significant bit
            assert(data_d[in_idx] <= UINT32_MAX / 2);
            assert(data_d[in_idx + 1] <= UINT32_MAX / 2);
            if (in_idx == 0) {
                inel2[0] = data_d[0];
            }
            else {
                inel2[0] = encoded_sign_diff(data_d[in_idx - 1], data_d[in_idx]);
            }
            inel2[1] = encoded_sign_diff(data_d[in_idx], data_d[in_idx + 1]);
        }
        else {
            inel2[0] = data_d[in_idx];
            inel2[1] = data_d[in_idx + 1];
        }
        for (int i = 0; i < 2; i++) {
            for (int j = 0; j < 4; j++) {
                // splice both u32 together bytewise
                working |= ((inel2[i] >> (j * 8)) & 0xFF) << ((j * 16) + (i * 8));
            }
            in_idx++;
        }
        // printf("WC:");
        // bit_print(working);
        // printf("\n");
        for (int i = 0; i < 9; i++) {
            // get rightmost 7 bits of working, mark as continued if working is non zero
            uint8_t acc;
            if (i == 8) {
                // if we need all 8 bytes, dont encode the continuation bit in the last one
                acc = (working & 0b11111111);
                working >>= 8;
            }
            else {
                acc = (working & 0b01111111);
                working >>= 7;
            }
            if (working) {
                acc |= 0b10000000;
            }
            data_c[out_byte_idx++] = acc;
            if (!working) {
                break;
            }
        }
    }
    return out_byte_idx;
}

template <size_t SB_ELEM_COUNT, bool USE_DIFFERENTIAL_PRE_ENCODING>
__host__ __device__ void decompress_block(size_t data_size, uint8_t* data_c, uint32_t* data_d)
{
    size_t in_byte_idx = 0;
    size_t out_idx = 0;
    uint32_t dbase;
    while (in_byte_idx < data_size) {
        uint64_t acc = 0;
        // accumulate bytes until one is marked as uncontinued
        uint8_t shift = 0;
        while (true) {
            assert(shift < 9);
            uint64_t working;
            if (shift == 8) {
                working = data_c[in_byte_idx++];
                acc |= (working << (shift++ * 7));
            }
            else {
                working = data_c[in_byte_idx++];
                acc |= ((working & 0b01111111) << (shift++ * 7));
            }
            if (!(working & 0b10000000) || shift == 9) {
                break;
            }
        }
        // printf("WD:");
        // bit_print(acc);
        // printf("\n");
        for (int i = 0; i < 2; i++) {
            uint32_t out_el = 0;
            for (int j = 0; j < 4; j++) {
                // unsplice into two u32 elements
                out_el |= ((acc >> ((j * 16) + (i * 8))) & 0xFF) << (j * 8);
            }
            if (USE_DIFFERENTIAL_PRE_ENCODING) {
                if (out_idx == 0) {
                    dbase = out_el;
                }
                else {
                    dbase = decode_sign_diff(dbase, out_el);
                    dbase &= 0b01111111111111111111111111111111;
                }
                data_d[out_idx++] = dbase;
            }
            else {
                data_d[out_idx++] = out_el;
            }
        }
    }
}

__device__ uint16_t ceil_size_to_elements(uint16_t size)
{
    return (size + sizeof(uint32_t) - 1) / sizeof(uint32_t) * sizeof(uint32_t);
}

template <size_t SB_ELEM_COUNT> __device__ void decompress_block_nop(size_t data_size, uint8_t* data_c, uint32_t* data_d)
{
    for (int i = 0; i < SB_ELEM_COUNT; i++) {
        data_d[i] = ((uint32_t*)data_c)[i];
    }
}

template <size_t SB_ELEM_COUNT> __device__ size_t compress_block_nop(uint32_t* data_d, uint8_t* data_c)
{
    for (int i = 0; i < SB_ELEM_COUNT; i++) {
        ((uint32_t*)data_c)[i] = data_d[i];
    }
    return SB_ELEM_COUNT * sizeof(uint32_t);
}

template <size_t SB_ELEM_COUNT> __device__ void decompress_sub_blocks(uint32_t* data_c, uint16_t* size_table, uint32_t* data_d)
{
    int warp_offset = get_warp_offset();
    uint32_t* begin_d = &data_d[warp_offset * SB_ELEM_COUNT];
    uint8_t* begin_c = (uint8_t*)&data_c[warp_offset * sb_max_elems_c(SB_ELEM_COUNT)];
    size_t prev_size = warp_offset == 0 ? 0 : ceil_size_to_elements(size_table[warp_offset - 1]);
    size_t size_c = size_table[warp_offset] - prev_size;
    decompress_block_nop<SB_ELEM_COUNT>(size_c, begin_c, begin_d);
    // decompress_block<SB_ELEM_COUNT, false>(size_c, begin_c, begin_d);
}

template <size_t SB_ELEM_COUNT> __device__ size_t compress_sub_blocks(uint32_t* data_d, uint16_t* size_table, uint32_t* data_c)
{
    int warp_offset = get_warp_offset();
    uint32_t* begin_d = &data_d[warp_offset * SB_ELEM_COUNT];
    uint8_t* begin_c = (uint8_t*)&data_c[warp_offset * sb_max_elems_c(SB_ELEM_COUNT)];
    return compress_block_nop<SB_ELEM_COUNT>(begin_d, begin_c);
    // return compress_block<SB_ELEM_COUNT, false>(begin_d, begin_c);
}

template <size_t SB_ELEM_COUNT> __device__ void build_size_table(uint16_t* size_table, uint16_t my_sb_size)
{
    int warp_offset = get_warp_offset();
    uint16_t ceiled_sb_size = (my_sb_size + sizeof(uint32_t) - 1) / sizeof(uint32_t) * sizeof(uint32_t);
    size_table[warp_offset] = ceiled_sb_size;
    __syncwarp();
    int i = 1;
    for (int i = 1; i < THREADS_PER_WARP; i *= 2) {
        if (warp_offset >= i) {
            size_table[warp_offset] += size_table[warp_offset - i];
        }
        __syncwarp();
        if (i == 0) break;
    }
    size_table[warp_offset] -= (ceiled_sb_size - my_sb_size);
}

template <size_t SB_ELEM_COUNT, uint32_t (*OPERATION)(uint32_t)>
__device__ void apply_operation(block_stats* b, uint32_t* data_d, uint16_t* size_table)
{
    int warp_offset = get_warp_offset();
    size_t sb_offset = warp_offset * SB_ELEM_COUNT;
    size_t sb_elem_count = SB_ELEM_COUNT;
    if (b->idx + 1 == b->count) {
        if (sb_offset >= b->final_block_elem_count) {
            sb_elem_count = 0;
        }
        else if (sb_offset + sb_elem_count > b->final_block_elem_count) {
            sb_elem_count = b->final_block_elem_count - sb_offset;
        }
    }

    uint32_t* sb_d_begin = (uint32_t*)&data_d[sb_offset];
    uint32_t* sb_d_end = (uint32_t*)&data_d[sb_offset + sb_elem_count];

    for (auto i = sb_d_begin; i != sb_d_end; i++) {
        *i = OPERATION(*i);
    }
}

template <size_t SB_ELEM_COUNT> __device__ void write_out_compressed_data(uint32_t* data_c, uint16_t* size_table, char* table_begin)
{
    int warp_offset = get_warp_offset();

    uint16_t size_val = size_table[warp_offset];
    ((uint16_t*)table_begin)[warp_offset] = size_val;
    size_table[warp_offset] = ceil_size_to_elements(size_val);
    __syncwarp();

    uint32_t* data_tgt = (uint32_t*)(table_begin + TABLE_SIZE);
    size_t data_offset = warp_offset * sizeof(uint32_t);
    int subblock_idx = warp_offset / sb_max_elems_c(SB_ELEM_COUNT);
    for (int i = 0; i < sb_max_elems_c(SB_ELEM_COUNT); i++) {
        for (;;) {
            if (subblock_idx == THREADS_PER_WARP) break;
            if (size_table[subblock_idx] > data_offset) break;
            subblock_idx++;
        }
        if (subblock_idx == THREADS_PER_WARP) break;
        size_t sb_offset = data_offset - (subblock_idx ? size_table[subblock_idx - 1] : 0);
        size_t data_c_idx = subblock_idx * sb_max_elems_c(SB_ELEM_COUNT) + sb_offset / sizeof(uint32_t);
        data_tgt[warp_offset + i * THREADS_PER_WARP] = data_c[data_c_idx];
        data_offset += THREADS_PER_WARP * sizeof(uint32_t);
    }
    __syncwarp();
    size_table[warp_offset] = size_val;
}

template <size_t SB_ELEM_COUNT> __device__ void load_in_compressed_data(char* table_begin, uint16_t* size_table, uint32_t* data_c)
{
    int warp_offset = get_warp_offset();

    uint16_t size_val = ((uint16_t*)table_begin)[warp_offset];
    size_table[warp_offset] = ceil_size_to_elements(size_val);
    __syncwarp();
    size_t data_offset = warp_offset * sizeof(uint32_t);
    int subblock_idx = warp_offset / sb_max_elems_c(SB_ELEM_COUNT);
    uint32_t* data_src = (uint32_t*)(table_begin + TABLE_SIZE);
    uint16_t raw_size_val = size_table[warp_offset];

    for (int i = 0; i < sb_max_elems_c(SB_ELEM_COUNT); i++) {
        for (;;) {
            if (subblock_idx == THREADS_PER_WARP) break;
            if (size_table[subblock_idx] > data_offset) break;
            subblock_idx++;
        }
        if (subblock_idx == THREADS_PER_WARP) break;
        size_t sb_offset = data_offset - (subblock_idx ? size_table[subblock_idx - 1] : 0);
        size_t data_c_idx = subblock_idx * sb_max_elems_c(SB_ELEM_COUNT) + sb_offset / sizeof(uint32_t);
        data_c[data_c_idx] = data_src[warp_offset + i * THREADS_PER_WARP];
        data_offset += THREADS_PER_WARP * sizeof(uint32_t);
    }
    __syncwarp();
    size_table[warp_offset] = size_val;
}

template <size_t SB_ELEM_COUNT> __device__ void load_in_uncompressed_data(uint32_t* data_src, uint32_t* data_d)
{
    int warp_offset = get_warp_offset();
    for (int i = warp_offset; i < SB_ELEM_COUNT * THREADS_PER_WARP; i += THREADS_PER_WARP) {
        data_d[i] = data_src[i];
    }
}

template <size_t SB_ELEM_COUNT> __device__ void write_out_uncompressed_data(block_stats* b, uint32_t* data_d, uint32_t* data_tgt)
{
    int warp_offset = get_warp_offset();
    size_t elem_count = SB_ELEM_COUNT * THREADS_PER_WARP;
    if (b->idx + 1 == b->count) {
        elem_count = b->final_block_elem_count;
    }

    for (int i = warp_offset; i < elem_count; i += THREADS_PER_WARP) {
        data_tgt[i] = data_d[i];
    }
}

template <int WARP_COUNT, size_t SB_ELEM_COUNT, uint32_t (*OPERATION)(uint32_t)>
__global__ void kernel_apply_unary_op_on_compressed(uint32_t* data_c, size_t uncompressed_data_size)
{
    __shared__ uint16_t size_table[WARP_COUNT][THREADS_PER_WARP * WARP_COUNT];
    __shared__ uint32_t s_mem_d[WARP_COUNT][SB_ELEM_COUNT * THREADS_PER_WARP];
    __shared__ uint32_t s_mem_c[WARP_COUNT][sb_max_elems_c(SB_ELEM_COUNT) * THREADS_PER_WARP];

    int warp_idx = get_warp_idx();
    block_stats b = get_block_stats(uncompressed_data_size, SB_ELEM_COUNT);

    for (; b.idx < b.count; b.idx += b.gridstride) {
        char* table_position = (char*)data_c + b.idx * b.size_c;
        load_in_compressed_data<SB_ELEM_COUNT>(table_position, size_table[warp_idx], s_mem_c[warp_idx]);

        decompress_sub_blocks<SB_ELEM_COUNT>(s_mem_c[warp_idx], size_table[warp_idx], s_mem_d[warp_idx]);

        apply_operation<SB_ELEM_COUNT, OPERATION>(&b, s_mem_d[warp_idx], size_table[warp_idx]);

        size_t sb_size = compress_sub_blocks<SB_ELEM_COUNT>(s_mem_d[warp_idx], size_table[warp_idx], s_mem_c[warp_idx]);
        build_size_table<SB_ELEM_COUNT>(size_table[warp_idx], sb_size);

        char* data_c_pos = (char*)data_c + b.idx * b.size_c;
        write_out_compressed_data<SB_ELEM_COUNT>(s_mem_c[warp_idx], size_table[warp_idx], data_c_pos);
    }
}

template <int WARP_COUNT, size_t SB_ELEM_COUNT>
__global__ void kernel_inital_compress(uint32_t* data_d, uint32_t* data_c, size_t uncompressed_data_size)
{
    __shared__ uint16_t size_table[WARP_COUNT][THREADS_PER_WARP];
    __shared__ uint32_t s_mem_d[WARP_COUNT][SB_ELEM_COUNT * THREADS_PER_WARP];
    __shared__ uint32_t s_mem_c[WARP_COUNT][sb_max_elems_c(SB_ELEM_COUNT) * THREADS_PER_WARP];

    int warp_idx = get_warp_idx();
    block_stats b = get_block_stats(uncompressed_data_size, SB_ELEM_COUNT);

    for (; b.idx < b.count; b.idx += b.gridstride) {
        char* initial_data_pos = (char*)data_d + b.idx * b.size_d;
        load_in_uncompressed_data<SB_ELEM_COUNT>((uint32_t*)initial_data_pos, s_mem_d[warp_idx]);

        size_t sb_size = compress_sub_blocks<SB_ELEM_COUNT>(s_mem_d[warp_idx], size_table[warp_idx], s_mem_c[warp_idx]);
        build_size_table<SB_ELEM_COUNT>(size_table[warp_idx], sb_size);

        char* data_c_pos = (char*)data_c + b.idx * b.size_c;
        write_out_compressed_data<SB_ELEM_COUNT>(s_mem_c[warp_idx], size_table[warp_idx], data_c_pos);
    }
}

template <int WARP_COUNT, size_t SB_ELEM_COUNT>
__global__ void kernel_final_decompress(uint32_t* data_c, uint32_t* data_d, size_t uncompressed_data_size)
{
    __shared__ uint16_t size_table[WARP_COUNT][THREADS_PER_WARP];
    __shared__ uint32_t s_mem_d[WARP_COUNT][SB_ELEM_COUNT * THREADS_PER_WARP];
    __shared__ uint32_t s_mem_c[WARP_COUNT][sb_max_elems_c(SB_ELEM_COUNT) * THREADS_PER_WARP];
    int warp_idx = get_warp_idx();
    block_stats b = get_block_stats(uncompressed_data_size, SB_ELEM_COUNT);
    for (; b.idx < b.count; b.idx += b.gridstride) {
        char* table_position = (char*)data_c + b.idx * b.size_c;

        load_in_compressed_data<SB_ELEM_COUNT>(table_position, size_table[warp_idx], s_mem_c[warp_idx]);

        decompress_sub_blocks<SB_ELEM_COUNT>(s_mem_c[warp_idx], size_table[warp_idx], s_mem_d[warp_idx]);

        char* data_d_pos = (char*)data_d + b.idx * b.size_d;
        write_out_uncompressed_data<SB_ELEM_COUNT>(&b, s_mem_d[warp_idx], (uint32_t*)data_d_pos);
    }
}

template <uint32_t (*OPERATION)(uint32_t)> __global__ void kernel_apply_unary_op_raw(uint32_t* data, size_t data_size)
{
    size_t grid_size = (size_t)blockDim.x * (size_t)gridDim.x;
    size_t pos = (size_t)threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = pos; i < data_size / sizeof(uint32_t); i += grid_size) {
        data[i] = OPERATION(data[i]);
    }
}

__device__ uint32_t inc(uint32_t x)
{
    return x + 1;
}

int main_foo()
{
    const size_t elements = /*1 << */ 20;
    size_t data_size = elements * sizeof(uint32_t);
    size_t worst_size = data_size + data_size / 8;
    uint32_t* in = (uint32_t*)malloc(data_size);
    uint32_t* res = (uint32_t*)malloc(data_size);
    uint8_t* out = (uint8_t*)malloc(worst_size);
    fast_prng rng(42);
    for (size_t i = 0; i < elements; i++) {
        uint32_t a = rng.rand();
        uint32_t b = rng.rand();
        uint32_t c = rng.rand();
        // TODO use some proper dataset to show all pros/cons of both algos

        // use this to show off normal ZS
        // in[i] = a % ((b >> (32 - (c & 0b11111))) + 1);

        // use this to show off basic diff enc
        // in[i] = i == 0 ? a : in[i - 1] + (a % 200);

        // use this to show off specifically the overflow diff enc
        if (i % 2 == 0) {
            in[i] = a | 0b01111111111111111111111110000000;
        }
        else {
            in[i] = a & 0b1111111;
        }

        in[i] &= 0b01111111111111111111111111111111; // sanitize for diff pre enc
    }
    // normal zero suppression
    size_t compressed_size = compress_block<elements, false>(in, out);
    printf("ZS compressed %zu data bytes into %zu / %zu\n", data_size, compressed_size, worst_size);
    printf("ZS ratio ZS: %.3f\n", (float)data_size / (float)compressed_size);
    decompress_block<elements, false>(compressed_size, out, res);
    for (size_t i = 0; i < elements; i++) {
        if (in[i] != res[i]) {
            printf("ZS FAIL @ %zu : I %u != O %u\n", i, in[i], res[i]);
            exit(1);
        }
    }
    printf("ZS PASS %zu\n", elements);
    // differential pre encoding into zero suppression
    compressed_size = compress_block<elements, true>(in, out);
    printf("DE-ZS compressed %zu data bytes into %zu / %zu\n", data_size, compressed_size, worst_size);
    printf("DE-ZS ratio: %.3f\n", (float)data_size / (float)compressed_size);
    decompress_block<elements, true>(compressed_size, out, res);
    for (size_t i = 0; i < elements; i++) {
        if (in[i] != res[i]) {
            printf("DE-ZS FAIL @ %zu : I %u != O %u\n", i, in[i], res[i]);
            exit(1);
        }
    }
    printf("DE-ZS PASS %zu\n", elements);
    free(in);
    free(out);
    free(res);
    return 0;
}

int main()
{
    constexpr size_t elem_count = 1 << 25;
    constexpr size_t sb_elem_count = 32;
    constexpr size_t warp_count = 4;
    constexpr size_t grid_size = 4096;

    constexpr size_t block_size = THREADS_PER_WARP * warp_count;
    size_t data_size = elem_count * sizeof(uint32_t);
    size_t data_size_compressed = get_compressed_size(data_size, elem_count);

    uint32_t* data_d = (uint32_t*)malloc(data_size);
    uint32_t* data_dest = (uint32_t*)malloc(data_size);

    uint32_t* data_c_gpu;
    uint32_t* data_d_gpu;
    uint32_t* data_res_gpu;
    CUDA_TRY(hipMalloc(&data_c_gpu, data_size_compressed));
    CUDA_TRY(hipMalloc(&data_d_gpu, data_size));
    CUDA_TRY(hipMalloc(&data_res_gpu, data_size));

    for (int i = 0; i < elem_count; i++) {
        data_d[i] = i;
    }
    memset(data_dest, 0, data_size);

    CUDA_TRY(hipMemcpy(data_d_gpu, data_d, data_size, hipMemcpyHostToDevice));

    float time_compressed = 0;
    float time_regular = 0;

    CUDA_QUICKTIME(&time_regular, kernel_apply_unary_op_raw<&inc><<<grid_size, block_size>>>(data_d_gpu, data_size););

    kernel_inital_compress<warp_count, sb_elem_count><<<grid_size, block_size>>>(data_d_gpu, data_c_gpu, data_size);
    CUDA_QUICKTIME(
        &time_compressed, kernel_apply_unary_op_on_compressed<warp_count, sb_elem_count, &inc><<<grid_size, block_size>>>(data_c_gpu, data_size));
    kernel_final_decompress<warp_count, sb_elem_count><<<grid_size, block_size>>>(data_c_gpu, data_res_gpu, data_size);

    CUDA_TRY(hipMemcpy(data_dest, data_res_gpu, data_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < elem_count; i++) {
        if (data_dest[i] != i + 2) {
            printf("fail: %i (+2): %u\n", i, data_dest[i]);
            return 1;
        }
    }
    printf("ok: regular: %f, compressed: %f\n", time_regular, time_compressed);
    return 0;
}
